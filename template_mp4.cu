#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define MASK_WIDTH 3
#define MASK_RADIUS 1
#define TILE_WIDTH 8
#define BLOCK_WIDTH (TILE_WIDTH + MASK_WIDTH - 1)
//@@ Define constant memory for device kernel here
__constant__ float M[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];

__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;

  int output_x = blockIdx.x * TILE_WIDTH + tx; //col_o
  int output_y = blockIdx.y * TILE_WIDTH + ty; //row_o
  int output_z = blockIdx.z * TILE_WIDTH + tz;

  int input_x = output_x - MASK_RADIUS; //col_i
  int input_y = output_y - MASK_RADIUS; //row_i
  int input_z = output_z - MASK_RADIUS;

  __shared__ float input_ds[BLOCK_WIDTH][BLOCK_WIDTH][BLOCK_WIDTH];

    if ((input_x >= 0) && (input_x < x_size) &&
        (input_y >= 0) && (input_y < y_size) &&
        (input_z >= 0) && (input_z < z_size))
    {
        input_ds[tz][ty][tx] = input[input_z * (y_size * x_size) + input_y * (x_size) + input_x];
    }
    else
    {
        input_ds[tz][ty][tx] = 0.0f;
    }
    __syncthreads();

    if (tx < TILE_WIDTH && ty < TILE_WIDTH && tz < TILE_WIDTH)
    {
        float Pvalue = 0;
        for (int i = 0; i < MASK_WIDTH; i++){
            for (int j = 0; j < MASK_WIDTH; j++){
                for (int k = 0; k < MASK_WIDTH; k++){
                    Pvalue += M[i][j][k] * input_ds[tz + i][ty + j][tx + k];
                }
            }
        }
        if (output_x < x_size && output_y < y_size && output_z < z_size)
        {
           output[output_z * (y_size * x_size) + output_y * (x_size) + output_x] = Pvalue;
        }
    }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions
  hipMalloc((void **)&deviceInput, (inputLength - 3) * sizeof(float));
  hipMalloc((void **)&deviceOutput, (inputLength - 3) * sizeof(float));

  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu
  hipMemcpy(deviceInput, &hostInput[3], (inputLength - 3) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(M), hostKernel, MASK_WIDTH * MASK_WIDTH * MASK_WIDTH * sizeof(float));
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Initialize grid and block dimensions here
  dim3 DimGrid(ceil(((float)x_size) / TILE_WIDTH), ceil(((float)y_size) / TILE_WIDTH), ceil(((float)z_size) / TILE_WIDTH));
  dim3 DimBlock(BLOCK_WIDTH, BLOCK_WIDTH, BLOCK_WIDTH);

  //@@ Launch the GPU kernel here
  conv3d<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, z_size, y_size, x_size);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  hipMemcpy(&hostOutput[3], deviceOutput, (inputLength - 3) * sizeof(float), hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  // Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}
